#define CATCH_CONFIG_MAIN
#include <hip/hip_runtime.h>

#include <SimElectronData.hh>
#include <SimGSTables.hh>
#include <SimIMFPBrem.hh>
#include <SimIMFPMaxPhoton.hh>
#include <SimIMFPMoller.hh>
#include <SimIMFPPhoton.hh>
#include <SimITr1MFPElastic.hh>
#include <SimKNTables.hh>
#include <SimMaterialData.hh>
#include <SimMaxScatStrength.hh>
#include <SimMollerTables.hh>
#include <SimPhotonData.hh>
#include <SimSBTables.hh>
#include <SimStoppingPower.hh>
#include <catch2/catch.hpp>
#include <cstdio>
#include <cuda/api.hpp>
#include <functional>
#include <random>

#include "gpu_unique_ptr.h"
#include "gpu_utils.h"
#include "test_utils.h"

namespace opmc {

static constexpr auto verbose_print = false;
static constexpr auto tests         = 16384;
static constexpr auto block_size    = 256;
static constexpr auto grid_size     = div_rounding_up(tests, block_size);

template <class F>
ODPM_KERNEL static void kernel(F f) {
    const auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < tests) {
        f(i);
    }
}

template <bool enable, typename... Ts>
static constexpr void debugPrintf(Ts... args) {
    if constexpr (enable) {
        printf(args...);
    }
}

#define INVOKE_TEST(arg)                    \
    kernel<<<grid_size, block_size>>>(arg); \
    gpuErrorAssert(hipDeviceSynchronize())

TEST_CASE("GPU INTERPOLATION TESTS") {
    const auto seed = std::random_device()();
    INFO("SEED " << seed);
    std::default_random_engine rng{seed};
    SimElectronData elData{};
    SimPhotonData phData{};
    SimMaterialData mData{};
    elData.Load(DATA_DIR, 0);
    phData.Load(DATA_DIR, 0);
    mData.Load(DATA_DIR, 0);
    TestMaterial material{DATA_DIR};
    TestPhotonData photonData{DATA_DIR};
    TestElectronData electronData{DATA_DIR};
    // This copies the data needed by the material on the GPU
    auto device = cuda::device::current::get();
    material.initializeGPU(device);
    photonData.initializeGPU(device);
    electronData.initializeGPU(device);
    // this allocates the material on the GPU and copies it
    auto gpu_material      = cuda::make_unique(device, material);
    auto gpu_electron_data = cuda::make_unique(device, electronData);
    auto gpu_photon_data   = cuda::make_unique(device, photonData);

    SECTION("ITr1MFPElastic") {
        std::uniform_real_distribution<real_type> distribution{material.GetITr1MFPElastic().getMinInput(),
                                                               material.GetITr1MFPElastic().getMaxInput()};

        std::vector<real_type> inputs(tests);
        std::generate(inputs.begin(), inputs.end(), [&distribution, &rng]() { return distribution(rng); });
        std::vector<real_type> reference(tests);
        std::transform(inputs.begin(), inputs.end(), reference.begin(), [&elData, &material](const real_type& x) {
            return elData.GetITr1MFPElastic()->GetITr1MFPPerDensity(x, material.id());
        });
        cuda::gpu_array<real_type> gpu_inputs(cuda::device::current::get(), inputs);
        cuda::gpu_array<real_type> gpu_outputs(cuda::device::current::get(), tests);
        auto gpu_function = [in = gpu_inputs.m_data(), out = gpu_outputs.m_data(),
                             material_ptr = gpu_material.get()] __device__(int i) {
            out[i] = material_ptr->iTr1MFPElasticElectron(in[i]);
            debugPrintf<verbose_print>("i = %d, in[i] = %f, out[i] = %f\n", i, in[i], out[i]);
        };

        INVOKE_TEST(gpu_function);

        auto gpu_results = gpu_outputs.to_vector();
        for (int i = 0; i < tests; ++i) {
            INFO("Test #: " << i)
            INFO("input: " << inputs[i])
            REQUIRE(reference[i] == Approx(gpu_results[i]));
        }
    }
    SECTION("MaxScatStrength") {
        std::uniform_real_distribution<real_type> distribution{material.GetMaxScatStrength().getMinInput(),
                                                               material.GetMaxScatStrength().getMaxInput()};

        std::vector<real_type> inputs(tests);
        std::generate(inputs.begin(), inputs.end(), [&distribution, &rng]() { return distribution(rng); });
        std::vector<real_type> reference(tests);
        std::transform(inputs.begin(), inputs.end(), reference.begin(), [&elData, &material](const real_type& x) {
            return elData.GetMaxScatStrength()->GetMaxScatStrength(x);
        });
        cuda::gpu_array<real_type> gpu_inputs(cuda::device::current::get(), inputs);
        cuda::gpu_array<real_type> gpu_outputs(cuda::device::current::get(), tests);
        auto gpu_function = [in = gpu_inputs.m_data(), out = gpu_outputs.m_data(),
                             material_ptr = gpu_material.get()] __device__(int i) {
            out[i] = material_ptr->maxScatteringStrengthElectron(in[i]);
            debugPrintf<verbose_print>("i = %d, in[i] = %f, out[i] = %f\n", i, in[i], out[i]);
        };

        INVOKE_TEST(gpu_function);

        auto gpu_results = gpu_outputs.to_vector();
        for (int i = 0; i < tests; ++i) {
            INFO("Test #: " << i)
            INFO("input: " << inputs[i])
            REQUIRE(reference[i] == Approx(gpu_results[i]));
        }
    }
    SECTION("IMFPMoller") {
        std::uniform_real_distribution<real_type> distribution{material.GetIMFPMoller().getMinInput(),
                                                               material.GetIMFPMoller().getMaxInput()};
        std::vector<real_type> inputs(tests);
        std::generate(inputs.begin(), inputs.end(), [&distribution, &rng]() { return distribution(rng); });
        std::vector<real_type> reference(tests);
        std::transform(inputs.begin(), inputs.end(), reference.begin(), [&elData, &material](const real_type& x) {
            return elData.GetIMFPMoller()->GetIMFPPerDensity(x);
        });
        cuda::gpu_array<real_type> gpu_inputs(cuda::device::current::get(), inputs);
        cuda::gpu_array<real_type> gpu_outputs(cuda::device::current::get(), tests);
        auto gpu_function = [in = gpu_inputs.m_data(), out = gpu_outputs.m_data(),
                             material_ptr = gpu_material.get()] __device__(int i) {
            out[i] = material_ptr->iMFPMollerElectron(in[i]);
            debugPrintf<verbose_print>("i = %d, in[i] = %f, out[i] = %f\n", i, in[i], out[i]);
        };

        INVOKE_TEST(gpu_function);

        auto gpu_results = gpu_outputs.to_vector();
        for (int i = 0; i < tests; ++i) {
            INFO("Test #: " << i)
            INFO("input: " << inputs[i])
            REQUIRE(reference[i] == Approx(gpu_results[i]));
        }
    }
    SECTION("IMFPBrem") {
        std::uniform_real_distribution<real_type> distribution{material.GetIMFPBrem().getMinInput(),
                                                               material.GetIMFPBrem().getMaxInput()};

        std::vector<real_type> inputs(tests);
        std::generate(inputs.begin(), inputs.end(), [&distribution, &rng]() { return distribution(rng); });
        std::vector<real_type> reference(tests);
        std::transform(inputs.begin(), inputs.end(), reference.begin(), [&elData, &material](const real_type& x) {
            return elData.GetIMFPBrem()->GetIMFPPerDensity(x, material.id());
        });
        cuda::gpu_array<real_type> gpu_inputs(cuda::device::current::get(), inputs);
        cuda::gpu_array<real_type> gpu_outputs(cuda::device::current::get(), tests);
        auto gpu_function = [in = gpu_inputs.m_data(), out = gpu_outputs.m_data(),
                             material_ptr = gpu_material.get()] __device__(int i) {
            out[i] = material_ptr->iMFPBremElectron(in[i]);
            debugPrintf<verbose_print>("i = %d, in[i] = %f, out[i] = %f\n", i, in[i], out[i]);
        };

        INVOKE_TEST(gpu_function);

        auto gpu_results = gpu_outputs.to_vector();
        for (int i = 0; i < tests; ++i) {
            INFO("Test #: " << i)
            INFO("input: " << inputs[i])
            REQUIRE(reference[i] == Approx(gpu_results[i]));
        }
    }
    SECTION("StoppingPower") {
        std::uniform_real_distribution<real_type> distribution{material.GetStoppingPower().getMinInput(),
                                                               material.GetStoppingPower().getMaxInput()};

        std::vector<real_type> inputs(tests);
        std::generate(inputs.begin(), inputs.end(), [&distribution, &rng]() { return distribution(rng); });
        std::vector<real_type> reference(tests);
        std::transform(inputs.begin(), inputs.end(), reference.begin(), [&elData, &material](const real_type& x) {
            return elData.GetDEDX()->GetDEDXPerDensity(x, material.id());
        });
        cuda::gpu_array<real_type> gpu_inputs(cuda::device::current::get(), inputs);
        cuda::gpu_array<real_type> gpu_outputs(cuda::device::current::get(), tests);
        auto gpu_function = [in = gpu_inputs.m_data(), out = gpu_outputs.m_data(),
                             material_ptr = gpu_material.get()] __device__(int i) {
            out[i] = material_ptr->stoppingPowerElectron(in[i]);
            debugPrintf<verbose_print>("i = %d, in[i] = %f, out[i] = %f\n", i, in[i], out[i]);
        };

        INVOKE_TEST(gpu_function);

        auto gpu_results = gpu_outputs.to_vector();
        for (int i = 0; i < tests; ++i) {
            INFO("Test #: " << i)
            INFO("input: " << inputs[i])
            REQUIRE(reference[i] == Approx(gpu_results[i]));
        }
    }
    SECTION("SeltzerBerger") {
        std::uniform_real_distribution<real_type> energy_distribution{constants::k_ElectronCut, constants::k_MaxEkin};
        std::uniform_real_distribution<real_type> uniform_distribution{};

        std::vector<real_type> inputs(tests), rng1(tests), rng2(tests), rng3(tests);
        std::generate(inputs.begin(), inputs.end(),
                      [&energy_distribution, &rng]() { return energy_distribution(rng); });
        std::generate(rng1.begin(), rng1.end(), [&uniform_distribution, &rng]() { return uniform_distribution(rng); });
        std::generate(rng2.begin(), rng2.end(), [&uniform_distribution, &rng]() { return uniform_distribution(rng); });
        std::generate(rng3.begin(), rng3.end(), [&uniform_distribution, &rng]() { return uniform_distribution(rng); });

        std::vector<real_type> reference(tests);
        for (int i = 0; i < tests; ++i) {
            reference[i] =
                elData.GetTheSBTables()->SampleEnergyTransfer(inputs[i], material.id(), rng1[i], rng2[i], rng3[i]);
        }
        cuda::gpu_array<real_type> gpu_inputs(cuda::device::current::get(), inputs);
        cuda::gpu_array<real_type> gpu_rng1(cuda::device::current::get(), rng1);
        cuda::gpu_array<real_type> gpu_rng2(cuda::device::current::get(), rng2);
        cuda::gpu_array<real_type> gpu_rng3(cuda::device::current::get(), rng3);
        cuda::gpu_array<real_type> gpu_outputs(cuda::device::current::get(), tests);

        auto gpu_function = [in = gpu_inputs.m_data(), out = gpu_outputs.m_data(), rng1 = gpu_rng1.m_data(),
                             rng2 = gpu_rng2.m_data(), rng3 = gpu_rng3.m_data(),
                             material_ptr = gpu_material.get()] __device__(int i) {
            out[i] = material_ptr->seltzerBergerElectron(in[i], rng1[i], rng2[i], rng3[i]);
            debugPrintf<verbose_print>("i = %d, in[i] = %f, out[i] = %f, rng1[i] = %f, rng2[i] = %f, rng3[i] = %f\n", i,
                                       in[i], out[i], rng1[i], rng2[i], rng3[i]);
        };

        INVOKE_TEST(gpu_function);

        auto gpu_results = gpu_outputs.to_vector();
        for (int i = 0; i < tests; ++i) {
            INFO("Test #: " << i)
            INFO("input: " << inputs[i])
            INFO("rng1 " << rng1[i])
            INFO("rng2 " << rng2[i])
            INFO("rng3 " << rng3[i])
            REQUIRE(reference[i] == Approx(gpu_results[i]));
        }
    }
    SECTION("MollerEnergyTransfer") {
        std::uniform_real_distribution<real_type> energy_distribution{constants::k_ElectronCut * 2,
                                                                      constants::k_MaxEkin};
        std::uniform_real_distribution<real_type> uniform_distribution{};

        std::vector<real_type> inputs(tests), rng1(tests), rng2(tests), rng3(tests);
        std::generate(inputs.begin(), inputs.end(),
                      [&energy_distribution, &rng]() { return energy_distribution(rng); });
        std::generate(rng1.begin(), rng1.end(), [&uniform_distribution, &rng]() { return uniform_distribution(rng); });
        std::generate(rng2.begin(), rng2.end(), [&uniform_distribution, &rng]() { return uniform_distribution(rng); });
        std::generate(rng3.begin(), rng3.end(), [&uniform_distribution, &rng]() { return uniform_distribution(rng); });

        std::vector<real_type> reference(tests);
        for (int i = 0; i < tests; ++i) {
            reference[i] = elData.GetTheMollerTables()->SampleEnergyTransfer(inputs[i], rng1[i], rng2[i], rng3[i]);
        }
        cuda::gpu_array<real_type> gpu_inputs(cuda::device::current::get(), inputs);
        cuda::gpu_array<real_type> gpu_rng1(cuda::device::current::get(), rng1);
        cuda::gpu_array<real_type> gpu_rng2(cuda::device::current::get(), rng2);
        cuda::gpu_array<real_type> gpu_rng3(cuda::device::current::get(), rng3);
        cuda::gpu_array<real_type> gpu_outputs(cuda::device::current::get(), tests);

        auto gpu_function = [in = gpu_inputs.m_data(), out = gpu_outputs.m_data(), rng1 = gpu_rng1.m_data(),
                             rng2 = gpu_rng2.m_data(), rng3 = gpu_rng3.m_data(),
                             electron_data_ptr = gpu_electron_data.get()] __device__(int i) {
            out[i] = electron_data_ptr->mollerEnergyTransfer(in[i], rng1[i], rng2[i], rng3[i]);
            debugPrintf<verbose_print>("i = %d, in[i] = %f, out[i] = %f, rng1[i] = %f, rng2[i] = %f, rng3[i] = %f\n", i,
                                       in[i], out[i], rng1[i], rng2[i], rng3[i]);
        };

        INVOKE_TEST(gpu_function);

        auto gpu_results = gpu_outputs.to_vector();
        for (int i = 0; i < tests; ++i) {
            INFO("Test #: " << i)
            INFO("input: " << inputs[i])
            INFO("rng1 " << rng1[i])
            INFO("rng2 " << rng2[i])
            INFO("rng3 " << rng3[i])
            REQUIRE(reference[i] == Approx(gpu_results[i]));
        }
    }
    SECTION("GoudsmitSaunderson") {
        std::uniform_real_distribution<real_type> energy_distribution{constants::k_ElectronCut, constants::k_MaxEkin};
        std::uniform_real_distribution<real_type> uniform_distribution{};

        std::vector<real_type> inputs(tests), rng1(tests), rng2(tests);
        std::generate(inputs.begin(), inputs.end(),
                      [&energy_distribution, &rng]() { return energy_distribution(rng); });
        std::generate(rng1.begin(), rng1.end(), [&uniform_distribution, &rng]() { return uniform_distribution(rng); });
        std::generate(rng2.begin(), rng2.end(), [&uniform_distribution, &rng]() { return uniform_distribution(rng); });

        std::vector<real_type> reference(tests);
        for (int i = 0; i < tests; ++i) {
            reference[i] = elData.GetTheGSTables()->SampleAngularDeflection(inputs[i], rng1[i], rng2[i]);
        }
        cuda::gpu_array<real_type> gpu_inputs(cuda::device::current::get(), inputs);
        cuda::gpu_array<real_type> gpu_rng1(cuda::device::current::get(), rng1);
        cuda::gpu_array<real_type> gpu_rng2(cuda::device::current::get(), rng2);
        cuda::gpu_array<real_type> gpu_outputs(cuda::device::current::get(), tests);
        auto gpu_function = [in = gpu_inputs.m_data(), out = gpu_outputs.m_data(), rng1 = gpu_rng1.m_data(),
                             rng2 = gpu_rng2.m_data(), material_ptr = gpu_material.get()] __device__(int i) {
            out[i] = material_ptr->angularDeflectionElectron(in[i], rng1[i], rng2[i]);
            debugPrintf<verbose_print>("i = %d, in[i] = %f, out[i] = %f, rng1[i] = %f, rng2[i] = %f\n", i, in[i],
                                       out[i], rng1[i], rng2[i]);
        };

        INVOKE_TEST(gpu_function);

        auto gpu_results = gpu_outputs.to_vector();
        for (int i = 0; i < tests; ++i) {
            INFO("Test #: " << i)
            INFO("input: " << inputs[i])
            INFO("rng1 " << rng1[i])
            INFO("rng2 " << rng2[i])
            REQUIRE(reference[i] == Approx(gpu_results[i]));
        }
    }

    SECTION("IMFPTotalPhoton") {
        std::uniform_real_distribution<real_type> distribution{material.GetIMFPTotalPhoton().GetMinInput(),
                                                               material.GetIMFPTotalPhoton().GetMaxInput()};

        std::vector<real_type> inputs(tests);
        std::generate(inputs.begin(), inputs.end(), [&distribution, &rng]() { return distribution(rng); });
        std::vector<real_type> reference(tests);
        std::transform(inputs.begin(), inputs.end(), reference.begin(), [&phData, &material](const real_type& x) {
            return phData.GetIMFPTotal()->GetIMFPPerDensity(x, material.id());
        });
        cuda::gpu_array<real_type> gpu_inputs(cuda::device::current::get(), inputs);
        cuda::gpu_array<real_type> gpu_outputs(cuda::device::current::get(), tests);
        auto gpu_function = [in = gpu_inputs.m_data(), out = gpu_outputs.m_data(),
                             material_ptr = gpu_material.get()] __device__(int i) {
            out[i] = material_ptr->iMFPTotalPhoton(in[i]);
            debugPrintf<verbose_print>("i = %d, in[i] = %f, out[i] = %f\n", i, in[i], out[i]);
        };

        INVOKE_TEST(gpu_function);

        auto gpu_results = gpu_outputs.to_vector();
        for (int i = 0; i < tests; ++i) {
            INFO("Test #: " << i)
            INFO("input: " << inputs[i])
            REQUIRE(reference[i] == Approx(gpu_results[i]));
        }
    }
    SECTION("IMFPComptonPhoton") {
        std::uniform_real_distribution<real_type> distribution{material.GetIMFPComptonPhoton().GetMinInput(),
                                                               material.GetIMFPComptonPhoton().GetMaxInput()};

        std::vector<real_type> inputs(tests);
        std::generate(inputs.begin(), inputs.end(), [&distribution, &rng]() { return distribution(rng); });
        std::vector<real_type> reference(tests);
        std::transform(inputs.begin(), inputs.end(), reference.begin(), [&phData, &material](const real_type& x) {
            return phData.GetIMFPCompton()->GetIMFPPerDensity(x, material.id());
        });
        cuda::gpu_array<real_type> gpu_inputs(cuda::device::current::get(), inputs);
        cuda::gpu_array<real_type> gpu_outputs(cuda::device::current::get(), tests);
        auto gpu_function = [in = gpu_inputs.m_data(), out = gpu_outputs.m_data(),
                             material_ptr = gpu_material.get()] __device__(int i) {
            out[i] = material_ptr->iMFPComptonPhoton(in[i]);
            debugPrintf<verbose_print>("i = %d, in[i] = %f, out[i] = %f\n", i, in[i], out[i]);
        };

        INVOKE_TEST(gpu_function);

        auto gpu_results = gpu_outputs.to_vector();
        for (int i = 0; i < tests; ++i) {
            INFO("Test #: " << i)
            INFO("input: " << inputs[i])
            REQUIRE(reference[i] == Approx(gpu_results[i]));
        }
    }
    SECTION("IMFPPairProdPhoton") {
        std::uniform_real_distribution<real_type> distribution{material.GetIMFPComptonPhoton().GetMinInput(),
                                                               material.GetIMFPComptonPhoton().GetMaxInput()};

        std::vector<real_type> inputs(tests);
        std::generate(inputs.begin(), inputs.end(), [&distribution, &rng]() { return distribution(rng); });
        std::vector<real_type> reference(tests);
        std::transform(inputs.begin(), inputs.end(), reference.begin(), [&phData, &material](const real_type& x) {
            return phData.GetIMFPCompton()->GetIMFPPerDensity(x, material.id());
        });
        cuda::gpu_array<real_type> gpu_inputs(cuda::device::current::get(), inputs);
        cuda::gpu_array<real_type> gpu_outputs(cuda::device::current::get(), tests);
        auto gpu_function = [in = gpu_inputs.m_data(), out = gpu_outputs.m_data(),
                             material_ptr = gpu_material.get()] __device__(int i) {
            out[i] = material_ptr->iMFPComptonPhoton(in[i]);
            debugPrintf<verbose_print>("i = %d, in[i] = %f, out[i] = %f\n", i, in[i], out[i]);
        };

        INVOKE_TEST(gpu_function);

        auto gpu_results = gpu_outputs.to_vector();
        for (int i = 0; i < tests; ++i) {
            INFO("Test #: " << i)
            INFO("input: " << inputs[i])
            REQUIRE(reference[i] == Approx(gpu_results[i]));
        }
    }
    SECTION("IMFPMaxPhoton") {
        std::uniform_real_distribution<real_type> distribution{photonData.GetIMFPMaxPhoton().GetMinInput(),
                                                               photonData.GetIMFPMaxPhoton().GetMaxInput()};

        std::vector<real_type> inputs(tests);
        std::generate(inputs.begin(), inputs.end(), [&distribution, &rng]() { return distribution(rng); });
        std::vector<real_type> reference(tests);
        std::transform(inputs.begin(), inputs.end(), reference.begin(),
                       [&phData](const real_type& x) { return phData.GetIMFPTotalMax()->GetIMFP(x); });
        cuda::gpu_array<real_type> gpu_inputs(cuda::device::current::get(), inputs);
        cuda::gpu_array<real_type> gpu_outputs(cuda::device::current::get(), tests);
        auto gpu_function = [in = gpu_inputs.m_data(), out = gpu_outputs.m_data(),
                             photon_data_ptr = gpu_photon_data.get()] __device__(int i) {
            out[i] = photon_data_ptr->iMFPMaxPhoton(in[i]);
            debugPrintf<verbose_print>("i = %d, in[i] = %f, out[i] = %f\n", i, in[i], out[i]);
        };

        INVOKE_TEST(gpu_function);

        auto gpu_results = gpu_outputs.to_vector();
        for (int i = 0; i < tests; ++i) {
            INFO("Test #: " << i)
            INFO("input: " << inputs[i])
            REQUIRE(reference[i] == Approx(gpu_results[i]));
        }
    }
    SECTION("KleinNishina") {
        std::uniform_real_distribution<real_type> energy_distribution{constants::k_GammaCut, constants::k_MaxEkin};
        std::uniform_real_distribution<real_type> uniform_distribution{};

        std::vector<real_type> inputs(tests), rng1(tests), rng2(tests), rng3(tests);
        std::generate(inputs.begin(), inputs.end(),
                      [&energy_distribution, &rng]() { return energy_distribution(rng); });
        std::generate(rng1.begin(), rng1.end(), [&uniform_distribution, &rng]() { return uniform_distribution(rng); });
        std::generate(rng2.begin(), rng2.end(), [&uniform_distribution, &rng]() { return uniform_distribution(rng); });
        std::generate(rng3.begin(), rng3.end(), [&uniform_distribution, &rng]() { return uniform_distribution(rng); });

        std::vector<real_type> reference(tests);
        for (int i = 0; i < tests; ++i) {
            reference[i] = phData.GetTheKNTables()->SampleEnergyTransfer(inputs[i], rng1[i], rng2[i], rng3[i]);
        }
        cuda::gpu_array<real_type> gpu_inputs(cuda::device::current::get(), inputs);
        cuda::gpu_array<real_type> gpu_rng1(cuda::device::current::get(), rng1);
        cuda::gpu_array<real_type> gpu_rng2(cuda::device::current::get(), rng2);
        cuda::gpu_array<real_type> gpu_rng3(cuda::device::current::get(), rng3);
        cuda::gpu_array<real_type> gpu_outputs(cuda::device::current::get(), tests);

        auto gpu_function = [in = gpu_inputs.m_data(), out = gpu_outputs.m_data(), rng1 = gpu_rng1.m_data(),
                             rng2 = gpu_rng2.m_data(), rng3 = gpu_rng3.m_data(),
                             photon_data_ptr = gpu_photon_data.get()] __device__(int i) {
            out[i] = photon_data_ptr->kleinNishina(in[i], rng1[i], rng2[i], rng3[i]);
            debugPrintf<verbose_print>("i = %d, in[i] = %f, out[i] = %f, rng1[i] = %f, rng2[i] = %f, rng3[i] = %f\n", i,
                                       in[i], out[i], rng1[i], rng2[i], rng3[i]);
        };

        INVOKE_TEST(gpu_function);

        auto gpu_results = gpu_outputs.to_vector();
        for (int i = 0; i < tests; ++i) {
            INFO("Test #: " << i)
            INFO("input: " << inputs[i])
            INFO("rng1 " << rng1[i])
            INFO("rng2 " << rng2[i])
            INFO("rng3 " << rng3[i])
            REQUIRE(reference[i] == Approx(gpu_results[i]));
        }
    }
}

}  // namespace opmc