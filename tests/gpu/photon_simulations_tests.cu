#include "hip/hip_runtime.h"
//
// Created by mbarbone on 5/30/22.
//
#define CATCH_CONFIG_MAIN
#include <Geom.hh>
#include <SimDPMLike.hh>
#include <SimMaterialData.hh>
#include <SimPhotonData.hh>
#include <Track.hh>
#include <catch2/catch.hpp>

#include "beams.h"
#include "constants.h"
#include "ksTest.h"
#include "physics.h"
#include "random.h"

namespace opmc {

constexpr auto voxel_size        = 2;
constexpr auto geom_size         = 200;
constexpr auto histories         = 1000192 * 4;
constexpr auto alpha             = 5.0E-8;
static constexpr auto block_size = 256;
static constexpr auto grid_size  = div_rounding_up(histories, block_size);

static ODPM_KERNEL void photonKernel(HalfDistanceVoxelCube* voxel_map, DummyQueue* queue, Water* material,
                                     PhotonData* photon_data, ElectronData* electron_data,
                                     const real_type primaryEnergy, const int gpu_histories, const unsigned seed) {
    Random rng{seed};
    Physics physics{rng, *voxel_map, *queue, *material, *photon_data, *electron_data};
    ThreeVector<real_type> initialPosition{0, 0, -0.5 * voxel_size};
    ThreeVector<real_type> initialDirection{0, 0, 1};
    PhotonPencilBeam beam{initialPosition, initialDirection, primaryEnergy};
    for (int i = 0; i < gpu_histories; ++i) {
        auto particle = beam.generateParticle(rng);
        physics.simulate(particle);
    }
}

TEST_CASE("GPU Photon simulation") {
    const unsigned seed = std::random_device()();
    INFO("SEED " << seed);
    SimMaterialData refMData;
    refMData.Load(DATA_DIR);
    SimPhotonData refPhData;
    refPhData.Load(DATA_DIR);
    auto device = cuda::device::current::get();
    ThreeVector<real_type> resolution(voxel_size, voxel_size, voxel_size);
    Water reference_material{DATA_DIR};
    reference_material.initializeGPU(device);
    auto gpu_material = cuda::make_unique(device, reference_material);

    PhotonData photon_data{DATA_DIR};
    photon_data.initializeGPU(device);
    auto gpu_photon_data = cuda::make_unique(device, photon_data);

    ElectronData electron_data{DATA_DIR};
    electron_data.initializeGPU(device);
    auto gpu_electron_data = cuda::make_unique(device, electron_data);

    DummyQueue queue;
    auto gpu_queue = cuda::make_unique(device, queue);

    for (int test_energy = 1; test_energy < 21; test_energy++) {
        SECTION("Energy " + std::to_string(test_energy) + "Mev") {
            const real_type primaryEnergy = test_energy;
            Geom geom(voxel_size, &refMData);
            Track primaryTrack;
            for (int i = 0; i < histories; ++i) {
                primaryTrack.fEkin         = primaryEnergy;  // Primary kinetic energy in [MeV]
                primaryTrack.fType         = 0;              // e-(-1) or photon(0)

                primaryTrack.fDirection[0] = 0.0;  // initial direction is [0,0,1]
                primaryTrack.fDirection[1] = 0.0;
                primaryTrack.fDirection[2] = 1.0;

                primaryTrack.fPosition[0]  = 0.0;  // initial position is [0,0, theRZ0]
                primaryTrack.fPosition[1]  = 0.0;
                primaryTrack.fPosition[2]  = -0.5 * voxel_size;
                KeepTrackingPhoton(refPhData, refMData, geom, primaryTrack);
            }

            auto reference = geom.histogram();
            while (!reference.empty() && reference[reference.size() - 1] == 0) {
                reference.pop_back();
            }
            reference.pop_back();

            real_type norm = 1. / *max_element(reference.begin(), reference.end());
            for (auto& item : reference) {
                item *= norm;
            }

            HalfDistanceVoxelCube voxel_map{{geom_size, geom_size, reference.size()}, resolution};
            voxel_map.setDefaultMaterial(gpu_material.get());
            voxel_map.initializeGPU(device);
            auto gpu_voxel_map = cuda::make_unique(device, voxel_map);

            cuda::launch(photonKernel, cuda::make_launch_config(grid_size, block_size), gpu_voxel_map.get(),
                         gpu_queue.get(), gpu_material.get(), gpu_photon_data.get(), gpu_electron_data.get(),
                         primaryEnergy, 1, seed);

            voxel_map.fromGPU();
            const auto result = voxel_map.doseDepthDistribution();
            for (auto& item : result) {
                std::cout << item << " ";
            }
            std::cout << std::endl;

            for (auto& item : reference) {
                std::cout << item << " ";
            }
            std::cout << std::endl;
            REQUIRE(opmc::KsTest(reference, result) >= alpha);
        }
        std::cout << std::endl;
    }
}
}  // namespace opmc